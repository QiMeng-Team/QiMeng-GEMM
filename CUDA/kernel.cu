#include <stdio.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>

void kernel_128_128(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C);
void kernel_64_128(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C);
void kernel_64_64(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C);
void kernel_32_64(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C);

void cuda_gemm(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C) {
    if (N >4096 || M > 4096) {
        kernel_128_128(M, N, K, alpha, A, B, beta, C);
    }
    else if (N >= 2048 || M>=2048) {
        kernel_64_128(M, N, K, alpha, A, B, beta, C);
    }
    else if (N==1024 && M==1024) {
        kernel_32_64(M, N, K, alpha, A, B, beta, C);
    }
    else {
        kernel_64_64(M, N, K, alpha, A, B, beta, C);
    }
}