#include <stdio.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include "cuda_kernel.cuh"

#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

void kernel_64_64(int M, int N, int K, float alpha, float *A, float *B,
                 float beta, float *C) {
    static const int BM = 64;
    static const int BN = 64;
    static const int BK = 16;
    static const int WM = 16;
    static const int WN = 32;
    static const int WMITER = 16;
    static const int WNITER = 32;
    static const int TM = 4;
    static const int TN = 4;

    dim3 threadsPerBlock((BM*BN)/(WM*WN)*32);
    dim3 blocksPerGrid(CEIL_DIV(N, BN), CEIL_DIV(M, BM));
    gemm<BM, BN, BK, WM, WN, WMITER, WNITER, TM, TN>
        <<<blocksPerGrid, threadsPerBlock>>>(M, N, K, alpha, A, B, beta, C);      
}